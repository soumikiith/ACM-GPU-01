#include <iostream>
#include <hip/hip_runtime.h>
__global__ void dkernel (){
	int a =2, b = 3;
	int c;
	c = a+b;
	int gid = threadIdx.x;
	printf("%d\n", gid); //still inside the size of printf buffer
}
int main(){
	dkernel <<<1,1024>>>();
	hipDeviceSynchronize();
	return 0;
}