#include <iostream>
#include <hip/hip_runtime.h>
__global__ void fun(int *a){
	int id = threadIdx.x;
	a[id] = id * id;
}
int main()
{
	int *array;
	int N=1024;
	array = (int*)calloc(sizeof(int),N);
	int *d_a;
	hipMalloc(&d_a,N*sizeof(int));
	fun<<<1,N>>>(d_a);
	hipDeviceSynchronize(); //can be commented also
	hipMemcpy(array, d_a, N*sizeof(int),hipMemcpyDeviceToHost);
	for(int i=0;i<N;i++){
		printf("%d\n", array[i]);
	}
	return 0;
}