#include <iostream>
#include <hip/hip_runtime.h>
__global__ void dkernel (){
	int a =2, b = 3;
	int c;
	c = a+b;
	printf("%d\n", c); //still inside the size of printf buffer
}
int main(){
	dkernel <<<1,32>>>();
	hipDeviceSynchronize();
	return 0;
}