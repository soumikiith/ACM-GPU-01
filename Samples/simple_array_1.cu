#include <iostream>
#include <hip/hip_runtime.h>
__global__ void fun(int *a){
	int id = threadIdx.x;
	a[id] = id * id;
}
int main()
{
	int *array;
	int N=100;
	array = (int*)calloc(sizeof(int),N);
	fun<<<1,N>>>(array);
	hipDeviceSynchronize();
	for(int i=0;i<N;i++){
		printf("%d\n", array[i]);
	}
	return 0;
}