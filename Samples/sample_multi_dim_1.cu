#include <iostream>
#include <hip/hip_runtime.h>
__global__ void dkernel (){
	printf("%d\n", threadIdx.x); //still inside the size of printf buffer
}
int main(){
	dim3 block(8,32,4);
	dkernel <<<2,128>>>();
	hipDeviceSynchronize();
	return 0;
}