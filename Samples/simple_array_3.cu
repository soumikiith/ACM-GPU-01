#include <iostream>
#include <hip/hip_runtime.h>
__global__ void fun(int *a){
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	a[id] = id;
}
int main()
{
	int *array;
	int N=2048;
	array = (int*)calloc(sizeof(int),N);
	int *d_a;
	hipMalloc(&d_a,N*sizeof(int));
	fun<<<2,1024>>>(d_a);
	hipDeviceSynchronize(); //can be commented also
	hipMemcpy(array, d_a, N*sizeof(int),hipMemcpyDeviceToHost);
	for(int i=0;i<N;i++){
		printf("%d\n", array[i]);
	}
	return 0;
}